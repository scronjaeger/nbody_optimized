#include "hip/hip_runtime.h"
#include "planets.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define RANDVEL 0.2f
#define GRAVITATION 0.02f

#define WIDTH 1366
#define HEIGHT 768

#define BLOCK_SIZE 256

#define SOFTENING 6.0
#define MAX(a, b) ((a) > (b) ? (a) : (b))

#define NUMPLANETS 64736

#define ALIGNEMENT 64

void CHECK_CUDA_ERR(hipError_t error)
{
    if(error != hipSuccess)
    {
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

extern "C"
{
    extern void glutSetOption(GLenum, int);
}
struct Planet
{
    float2 position;
    float2 velocity;
    float2 acceleration;
};

static struct Planet *host_planets;
static struct Planet *device_planets;

static dim3 block(BLOCK_SIZE);
static dim3 grid((NUMPLANETS + BLOCK_SIZE - 1) / BLOCK_SIZE);

void init(void)
{
    for(int i = 0; i < NUMPLANETS; i++)
    {
        host_planets[i].position.x = rand() % WIDTH;
        host_planets[i].position.y = rand() % HEIGHT;
        float x_diff               = host_planets[i].position.x - WIDTH / 2.f;
        float y_diff               = host_planets[i].position.y - HEIGHT / 2.f;

        float rand_x = (float)(rand()) / (float)RAND_MAX * (RANDVEL * 2) - RANDVEL;
        float rand_y = (float)(rand()) / (float)RAND_MAX * (RANDVEL * 2) - RANDVEL;

        float distance             = sqrtf(x_diff * x_diff + y_diff * y_diff);
        host_planets[i].velocity.x = -y_diff / sqrtf(distance) * 2 * GRAVITATION + rand_x;
        host_planets[i].velocity.y = x_diff / sqrtf(distance) * 2 * GRAVITATION + rand_y;
    }
}

__global__ void calcForces(struct Planet *planets)
{
    int i = (int)(blockIdx.x * blockDim.x + threadIdx.x);
    if(i < NUMPLANETS)
    {
        float2 force = make_float2(0.f, 0.f);
        for(int j = 0; j < NUMPLANETS; j++)
        {
            if(i != j)
            {
                float2 direction;
                direction.x = planets[j].position.x - planets[i].position.x;
                direction.y = planets[j].position.y - planets[i].position.y;

                float distsqr = (direction.x * direction.x) + (direction.y * direction.y) + SOFTENING;
                float distinv = 1 / sqrtf(distsqr);

                force.x += distinv * distinv * distinv * direction.x * GRAVITATION;
                force.y += distinv * distinv * distinv * direction.y * GRAVITATION;
            }
        }
        planets[i].acceleration = force;
    }
}

__global__ void updateVelPos(struct Planet *planets)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < NUMPLANETS)
    {
        planets[i].velocity.x += planets[i].acceleration.x;
        planets[i].velocity.y += planets[i].acceleration.y;
        planets[i].position.x += planets[i].velocity.x;
        planets[i].position.y += planets[i].velocity.y;
    }
}

#ifdef VISUALIZE

static char keyStates[256];

void draw_planet(void)
{
    glColor3f(1.0f, 1.0f, 1.0f);
    glPointSize(1);
    glBegin(GL_POINTS);
    for(int i = 0; i < NUMPLANETS; i++)
    {
        glVertex2f(host_planets[i].position.x, host_planets[i].position.y);
    }
    glEnd();
}

void Initialize(void)
{
    glClearColor(0, 0, 0, 1.0);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, WIDTH, 0.0, HEIGHT, 0.0, 1.0);
}

void keyOperations(void)
{
    if(keyStates[114])
    {
        init();
        CHECK_CUDA_ERR(
            hipMemcpy(device_planets, host_planets, NUMPLANETS * sizeof(struct Planet), hipMemcpyHostToDevice));

        keyStates[114] = 0;
    }
    if(keyStates[113] || keyStates[27])
    {
        exit(0);
    }
}

void keyPressed(unsigned char key, int x, int y)
{
    keyStates[key] = 1;
}

void render(void)
{
    CHECK_CUDA_ERR(
        hipMemcpy(host_planets, device_planets, NUMPLANETS * sizeof(struct Planet), hipMemcpyDeviceToHost));
    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glClear(GL_COLOR_BUFFER_BIT);
    draw_planet();
    glutSwapBuffers();
    keyOperations();
}

void update(void)
{
    calcForces<<<grid, block>>>(device_planets);
    CHECK_CUDA_ERR(hipGetLastError());

    updateVelPos<<<grid, block>>>(device_planets);
    CHECK_CUDA_ERR(hipGetLastError());

    CHECK_CUDA_ERR(hipDeviceSynchronize());
}

void timer(int value)
{
    update();

    glutTimerFunc(0, timer, 0);
    glutPostRedisplay();
}

int main(int argc, char *argv[])
{
    host_planets = (struct Planet *)malloc(sizeof(struct Planet) * NUMPLANETS);
    init();
    CHECK_CUDA_ERR(hipMalloc((void **)&device_planets, sizeof(struct Planet) * NUMPLANETS));
    CHECK_CUDA_ERR(
        hipMemcpy(device_planets, host_planets, NUMPLANETS * sizeof(struct Planet), hipMemcpyHostToDevice));

    glutInit(&argc, argv);
    glutSetOption(GLUT_MULTISAMPLE, 8);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_MULTISAMPLE);
    glEnable(GL_MULTISAMPLE);
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("Planets");
    glutFullScreen();
    Initialize();
    glutDisplayFunc(&render);
    glutKeyboardFunc(keyPressed);
    glutIdleFunc(NULL);

    glewInit();
    if(!GLEW_VERSION_2_0)
    {
        fprintf(stderr, "OpenGL 2.0 not available\n");
        return 1;
    }

    glutTimerFunc(0, timer, 0);
    glutMainLoop();

    return 0;
}
#else
int main(void)
{
    struct timespec start, end;

    time_t t;
    srand((unsigned)time(&t));

    host_planets = (struct Planet *)malloc(sizeof(struct Planet) * NUMPLANETS);
    CHECK_CUDA_ERR(hipMalloc((void **)&device_planets, sizeof(struct Planet) * NUMPLANETS));
    init();
    CHECK_CUDA_ERR(
        hipMemcpy(device_planets, host_planets, NUMPLANETS * sizeof(struct Planet), hipMemcpyHostToDevice));

    int interval = 1;
    double elapsed;
    for(;;)
    {
        clock_gettime(CLOCK_MONOTONIC, &start);
        for(int i = 0; i < interval; i++)
        {
            calcForces<<<grid, block>>>(device_planets);
            CHECK_CUDA_ERR(hipGetLastError());
            CHECK_CUDA_ERR(hipDeviceSynchronize());
            updateVelPos<<<grid, block>>>(device_planets);
            CHECK_CUDA_ERR(hipGetLastError());
            CHECK_CUDA_ERR(hipDeviceSynchronize());
        }
        clock_gettime(CLOCK_MONOTONIC, &end);

        elapsed = (double)(end.tv_sec - start.tv_sec) + (double)(end.tv_nsec - start.tv_nsec) / 1e9;

        double gens = interval / elapsed;
        interval    = MAX((int)(interval / elapsed), 1);
        printf("%.2e g/s | %.2e OPS\n", gens, gens * NUMPLANETS * NUMPLANETS);
        fflush(stdout);
    }
    free(host_planets);
    CHECK_CUDA_ERR(hipFree(device_planets));
}
#endif
